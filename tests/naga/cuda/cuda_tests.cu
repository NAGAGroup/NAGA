#include "hip/hip_runtime.h"
// Copyright 2023 Jack Myers
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

//
// Created by jack on 1/12/23.
//

#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "doctest.h"
#include <naga/cuda/cuda.cuh>

TEST_CASE("naga::cuda::cuda_error") {
    CHECK(naga::cuda::cuda_error(hipSuccess).success());
    CHECK(!naga::cuda::cuda_error(hipErrorOutOfMemory).success());

    naga::cuda::cuda_error error(hipErrorOutOfMemory);
    CHECK_THROWS_AS(error.raise_if_error(), naga::cuda::cuda_exception);
    CHECK(error.to_string() == "out of memory");
    CHECK(error.get() == hipErrorOutOfMemory);

    CHECK(naga::cuda::get_last_error().success());

    hipMalloc(nullptr, 0);
    error = naga::cuda::peek_last_error();
    CHECK(!error.success());
    CHECK_THROWS_AS(error.raise_if_error(), naga::cuda::cuda_exception);
    CHECK(error.to_string() == "invalid argument");

    error = naga::cuda::get_last_error();
    CHECK(!error.success());
    CHECK_THROWS_AS(error.raise_if_error(), naga::cuda::cuda_exception);
    CHECK(error.to_string() == "invalid argument");

    error = naga::cuda::get_last_error();
    CHECK(error.success());
    CHECK_NOTHROW(error.raise_if_error());
}

TEST_CASE("naga::cuda::context_manager") {
    using context_manager = naga::cuda::runtime;

    float* device_ptr;
    size_t available, total;
    hipMemGetInfo(&available, &total);
    hipMalloc(&device_ptr, available / 3);
    context_manager::system_reset();
    size_t available_after_reset, total_after_reset;
    hipMemGetInfo(&available_after_reset, &total_after_reset);
    CHECK(available == available_after_reset);

    int device_count = 0;
    hipGetDeviceCount(&device_count);
    CHECK(naga::cuda::runtime::get_device_count() == device_count);

    CHECK_THROWS_AS(
        context_manager::set_device(100),
        naga::cuda::cuda_exception
    );
    CHECK_THROWS_WITH(
        context_manager::set_device(100),
        "static int naga::cuda::context_manager::set_device(int) failed with "
        "error: invalid device ordinal"
    );

    CHECK_NOTHROW(context_manager::set_device(0));
    CHECK(context_manager::get_device() == 0);

    CHECK_THROWS_AS(
        context_manager::set_device(context_manager::cpu_device_id),
        std::invalid_argument
    );
    CHECK_THROWS_WITH(
        context_manager::set_device(context_manager::cpu_device_id),
        "static int naga::cuda::context_manager::set_device(int): "
        "cpu_device_id and distributed_device_id are not valid device ids"
    );

    CHECK_THROWS_AS(
        context_manager::set_device(context_manager::distributed_device_id),
        std::invalid_argument
    );
    CHECK_THROWS_WITH(
        context_manager::set_device(context_manager::distributed_device_id),
        "static int naga::cuda::context_manager::set_device(int): "
        "cpu_device_id and distributed_device_id are not valid device ids"
    );
}

__global__ void set_array_kernel(int* array, int value, size_t size) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size;
         i += gridDim.x * blockDim.x) {
        //        printf("setting array[%u] = %u\n", static_cast<unsigned
        //        int>(i), value);
        array[i] = value;
    }
}

TEST_CASE("naga::cuda::synchronize") {
    int* unified_ptr;
    uint num_elements = 1e6;
    hipMallocManaged(&unified_ptr, num_elements * sizeof(int));

    std::fill_n(unified_ptr, num_elements, 0);

    set_array_kernel<<<1, 1>>>(unified_ptr, 1, num_elements);
    naga::cuda::synchronize();

    CHECK(std::all_of(unified_ptr, unified_ptr + num_elements, [](int i) {
        return i == 1;
    }));
}

__global__ void set_array_kernel_distributed(
    int,
    int dev_id,
    size_t device_problem_size,
    int* array,
    int value,
    size_t size
) {
    for (size_t i
         = blockIdx.x * blockDim.x + threadIdx.x + dev_id * device_problem_size;
         i < size && i < (dev_id + 1) * device_problem_size;
         i += gridDim.x * blockDim.x) {
        //        printf("setting array[%u] = %u\n", static_cast<unsigned
        //        int>(i), value);
        array[i] = value;
    }
}

auto distributed_kernel_launch = naga::cuda::
    distributed_kernel_launch<set_array_kernel_distributed, int*, int, size_t>;

auto kernel_launch
    = naga::cuda::kernel_launch<set_array_kernel, int*, int, size_t>;

TEST_CASE("naga::cuda::*kernel*_launch") {
    int* unified_ptr;
    uint num_elements = 1e6;
    hipMallocManaged(&unified_ptr, num_elements * sizeof(int));
    std::fill_n(unified_ptr, num_elements, 0);

    using naga::cuda::execution_policy;
    auto stream = kernel_launch(
        execution_policy::async,
        1,
        1,
        0,
        0,
        unified_ptr,
        1,
        num_elements
    );

    CHECK(!std::all_of(unified_ptr, unified_ptr + num_elements, [](int i) {
        return i == 1;
    }));

    stream.synchronize();

    CHECK(std::all_of(unified_ptr, unified_ptr + num_elements, [](int i) {
        return i == 1;
    }));

    CHECK(stream.get() != hipStreamDefault);

    auto streams = distributed_kernel_launch(
        execution_policy::sync,
        num_elements,
        1,
        1,
        0,
        unified_ptr,
        2,
        num_elements
    );

    CHECK(std::all_of(unified_ptr, unified_ptr + num_elements, [](int i) {
        return i == 2;
    }));
}